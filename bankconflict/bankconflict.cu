#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "utils.hpp"

#define N 1024 // Size of the array

#define CHECK_CUDA_ERROR(func) \
    do { \
        hipError_t error = (func); \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << ": " \
                      << hipGetErrorString(error) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while(0)


__global__ void kernel(int* array, int stride, int* out) {
    __shared__ int sharedArray[N * 12];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread loads its corresponding element into shared memory
    sharedArray[tid] = array[tid];

    __syncthreads();

    // Simulate a read operation from shared memory
    out[tid] = sharedArray[tid] + sharedArray[tid + stride];
}

int main(int argc, char** argv) {
    int h_in[N];
    int h_out[N];
    auto [stride, iteration] = parseCommandLineArguments(argc, argv);

    printf("begin with stride=%d, iteration=%d\n", stride, iteration);

    // Initialize array data
    for (int i = 0; i < N; ++i) {
        h_in[i] = 1;
    }

    int *d_in, *d_out;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_in, N * sizeof(int)));
    CHECK_CUDA_ERROR(hipMemcpy(d_in, h_in, N * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_out, N * sizeof(int)));

    dim3 threadsPerBlock(N);
    dim3 numBlocks(1, 1);

    hipEvent_t start, end;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&end));

    CHECK_CUDA_ERROR(hipEventRecord(start));
    kernel<<<numBlocks, threadsPerBlock>>>(d_in, stride, d_out);
    for (int i = 0; i < iteration; i++){
        hipError_t err = hipGetLastError();

        if (err != hipSuccess){
            fprintf(stderr, "Failed to launch convert kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
    CHECK_CUDA_ERROR(hipMemcpy((void*)h_out, (void*)d_out, N * sizeof(int), hipMemcpyDeviceToHost));

    CHECK_CUDA_ERROR(hipEventRecord(end));
    CHECK_CUDA_ERROR(hipEventSynchronize(end));

    // for(int i=0; i<N; i++)
    // {
    //     printf("h_out[%d] = %d\n", i, h_out[i]);
    // }
    float milliseconds = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, end));
    printf("kernel excution time total=%f, avg=%f ms\n", milliseconds, milliseconds/iteration);

    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(end));

    CHECK_CUDA_ERROR(hipFree(d_in));

    return 0;
}
