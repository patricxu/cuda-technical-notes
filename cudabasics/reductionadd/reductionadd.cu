#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>
#include <chrono> // For timing
#include "utils.hpp"
// #include "reductionadd.hpp"


__global__ void reduce0(int *g_idata, int inLen, int *g_odata) {
    extern __shared__ int sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0){
        g_odata[blockIdx.x] = sdata[0];
    }
}


__global__ void reduce1(int *g_idata, int inLen, int *g_odata) {
    extern __shared__ int sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        int index = s * tid * 2;
        if (index < blockDim.x) {
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}


__global__ void reduce2(int *g_idata, int inLen, int *g_odata) {
    extern __shared__ int sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(tid < s)
            sdata[tid] += sdata[tid + s];
        
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}


__global__ void reduce3(int *g_idata, int inLen, int *g_odata) {
    extern __shared__ int sdata[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x / 2];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s = blockDim.x / 4; s > 0; s >>= 1) {
        if(tid < s)
            sdata[tid] += sdata[tid + s];
        
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0) 
    {
        g_odata[blockIdx.x] = sdata[0];
    }
}


__global__ void reduce4(int *g_idata, int inLen, int *g_odata) {
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load element from global memory to shared memory
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x / 2];
    __syncthreads();

    // Perform reduction in shared memory using a binary tree approach
    #pragma unroll
    for (unsigned int s = N / 4; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}


__device__ void warpReduce(volatile int* sdata, unsigned int tid) {
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1]; 
}

__global__ void reduce5(int *g_idata, int inLen, int *g_odata) {
    extern __shared__  int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load element from global memory to shared memory
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x / 2];
    __syncthreads();

    // Perform reduction in shared memory using a binary tree approach
    #pragma unroll
    for (unsigned int s = N / 4; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid < 32) {
        warpReduce(sdata, tid);
    }
    // Write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}


template <unsigned int blockSize>
__device__ void warpReduce1(volatile int* sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >=2) sdata[tid] += sdata[tid + 1]; 
}


template <unsigned int blockSize>
__global__ void reduce6(int *g_idata, int inLen, int *g_odata) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i + blockDim.x / 2];
    __syncthreads();

    if (blockSize > 512) {
        if (tid < 256) {
            sdata[tid] += sdata[tid + 256];
            __syncthreads();
        }
    }

    if (blockSize > 256) {
        if (tid < 128) {
            sdata[tid] += sdata[tid + 128];
            __syncthreads();
        }
    }

    if (blockSize > 128) {
        if (tid < 64) {
            sdata[tid] += sdata[tid + 64];
            __syncthreads();
        }
    }

    if (tid < 32) {
        warpReduce1<blockSize>(sdata, tid);
    }

    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}

int *d_in_extend = nullptr;

void callKernel(int reductioinNum, int *d_in, int inLen, int *d_out, dim3 blockDim, dim3 threadDim) {
    // printf("call kernel reductioinNum=%d, inLen=%d, blockDim.x=%d, threadDim.x=%d\n", reductioinNum, inLen, blockDim.x, threadDim.x);

    if (d_in_extend == nullptr){
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_in_extend, N * sizeof(int)));
    }

    if (inLen < N){
        CHECK_CUDA_ERROR(hipMemset(d_in_extend, 0, N * sizeof(int)));
        CHECK_CUDA_ERROR(hipMemcpy((void*)d_in_extend, d_in, inLen * sizeof(int), hipMemcpyDeviceToDevice));
    }

    switch (reductioinNum)
        {
        case 0:
            reduce0<<<blockDim, threadDim, N * sizeof(int)>>>(inLen < N ? d_in_extend : d_in, inLen, d_out);
            break;
        case 1:
            reduce1<<<blockDim, threadDim, N * sizeof(int)>>>(inLen < N ? d_in_extend : d_in, inLen, d_out);
            break;
        case 2:
            reduce2<<<blockDim, threadDim, N * sizeof(int)>>>(inLen < N ? d_in_extend : d_in, inLen, d_out);
            break;
        case 3:
            reduce3<<<blockDim, threadDim, N * sizeof(int)>>>(inLen < N ? d_in_extend : d_in, inLen, d_out);
            break;
        case 4:
            reduce4<<<blockDim, threadDim, N * sizeof(int)>>>(inLen < N ? d_in_extend : d_in, inLen, d_out);
            break;
        case 5:
            reduce5<<<blockDim, threadDim, N * sizeof(int)>>>(inLen < N ? d_in_extend : d_in, inLen, d_out);
            break;
        case 6:
            reduce6<N><<<blockDim, threadDim, N * sizeof(int)>>>(inLen < N ? d_in_extend : d_in, inLen, d_out);
        default:
            break;
    }

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();

    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch convert kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }    
}


int main(int argc, char** argv) {
    int *h_in;
    int *h_out;
    int *d_in, *d_out;
    int *d_partialBuff, *d_partialSum;
    int dPartialBuffLen = 0;
    int dArraySize = 0;
    auto [reductioinNum, arraySize] = parseCommandLineArguments(argc, argv);

    printf("begin with reduction%d, arraySize=%d\n", reductioinNum, arraySize);

    h_in = (int*)malloc(arraySize * sizeof(int));

    // Initialize array data
    for (int i = 0; i < arraySize; ++i) {
        h_in[i] = 1;
    }

    cpuReduce(h_in, arraySize);

    h_out = (int*)malloc(sizeof(int));

    // Adjest the device array size
    for (int mask = (1 << INT32_WIDTH - 2), i = 0; mask > 0; mask >>= 1, i++) {
        if (arraySize & mask){
            if (mask < N) {
                dArraySize += N;
                dPartialBuffLen += 1;
                break;
            }
            else{
                dPartialBuffLen += mask / N;
                dArraySize += mask;
            }
        }
    }

    printf("dArraySize=%d, dPartialBuffLen=%d\n", dArraySize, dPartialBuffLen);
    
    dim3 threadDim(N);
    dim3 blockDim_1(1);
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_in, dArraySize * sizeof(int)));
    CHECK_CUDA_ERROR(hipMemset(d_in, 0, dArraySize * sizeof(int)));
    CHECK_CUDA_ERROR(hipMemcpy(d_in, h_in, arraySize * sizeof(int), hipMemcpyHostToDevice));

    CHECK_CUDA_ERROR(hipMalloc((void**)&d_out, sizeof(int)));
    CHECK_CUDA_ERROR(hipMemset(d_out, 0, sizeof(int)));

    CHECK_CUDA_ERROR(hipMalloc((void**)&d_partialBuff, dPartialBuffLen * sizeof(int)));
    CHECK_CUDA_ERROR(hipMemset(d_partialBuff, 0, dPartialBuffLen * sizeof(int)));

    CHECK_CUDA_ERROR(hipMalloc((void**)&d_partialSum, INT32_WIDTH * sizeof(int)));
    CHECK_CUDA_ERROR(hipMemset(d_partialSum, 0, INT32_WIDTH * sizeof(int)));

    hipEvent_t start, end;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&end));
    CHECK_CUDA_ERROR(hipEventRecord(start));

    for (int mask = (1 << INT32_WIDTH - 2), *d_partialBuffCurosr = d_partialBuff, *d_inCursor = d_in; mask > 0; mask >>= 1) {
        if (dArraySize & mask) {
            if (mask < N) {
                //launch the kernel with coalsed input data. should be the final kernel call
                callKernel(reductioinNum, d_inCursor, N, &d_partialSum[(int)log2(N)], blockDim_1, threadDim);
                break;
            }
            else{
                dim3 blockDim(mask / N);
                int * d_tmp_in = d_inCursor;
                int * d_tmp_out = d_partialBuffCurosr;
                int * d_paritalSum_out;
                for(int j = mask; j > 1; j /= N){
                    callKernel(reductioinNum, d_tmp_in, j, d_tmp_out, blockDim, threadDim);
                    d_paritalSum_out = d_tmp_out;
                    d_tmp_out = d_tmp_in;
                    d_tmp_in = d_paritalSum_out;
                    blockDim.x = blockDim.x / threadDim.x + 1;
                }
                int h_paritalSum_out;
                CHECK_CUDA_ERROR(hipMemcpy((void*)&h_paritalSum_out, d_paritalSum_out, sizeof(int), hipMemcpyDeviceToHost));
                // printf("h_paritalSum_out=%d\n", h_paritalSum_out);
                CHECK_CUDA_ERROR(hipMemcpy((void*)&d_partialSum[(int)log2(mask)], d_paritalSum_out, sizeof(int), hipMemcpyDeviceToDevice));
                d_inCursor += mask;
                d_partialBuffCurosr += mask / N;
            }
        }
    }

    callKernel(reductioinNum, d_partialSum, INT32_WIDTH, d_out, blockDim_1, threadDim);
    CHECK_CUDA_ERROR(hipMemcpy((void*)h_out, (void*)d_out, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipEventRecord(end));
    CHECK_CUDA_ERROR(hipEventSynchronize(end));

    float milliseconds = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, end));
    printf("sum=%d GPU execution time total=%fms\n", *h_out, milliseconds);

    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(end));
    
    CHECK_CUDA_ERROR(hipFree(d_partialSum));
    CHECK_CUDA_ERROR(hipFree(d_partialBuff));
    CHECK_CUDA_ERROR(hipFree(d_in));
    CHECK_CUDA_ERROR(hipFree(d_out));

    if (d_in_extend != nullptr)
        CHECK_CUDA_ERROR(hipFree(d_in_extend));

    free(h_in);
    free(h_out);

    return 0;
}

